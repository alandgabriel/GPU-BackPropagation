#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include<fstream>
#include <sstream>
#include <vector>
#define HX 558 //Filas de X  train:558, total:699
#define WX 9 //Columnas de X
#define HY 558 //Filas de Y train:558, total:699
#define HXtest 141
#define HYtest 141
#define FILE_NAME_TRAIN "/home/alan/Documents/GPUs_Deeplearning_IIMAS/Cuda NNS/breast-cancer-train.csv"
#define FILE_NAME_TEST "/home/alan/Documents/GPUs_Deeplearning_IIMAS/Cuda NNS/breast-cancer-test.csv"
#define NUM_ITER 2000
#define alphamax 0.5
#define NUM_MODELOS 100
#define TILE_W 10 //Tamaño mosaicos
#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__));}
using namespace std;

void MatMul(float *a,float *d,float *c,int ha,int wa,int wb);
void generateRandom(float *h_a,int rows, int cols);
void leerDatos(float *X,float *y,string file);
void printMatrix(float *h_a, int rows, int cols);
void sigmoid(float *a,float *c,int h);
float costoCE(float *a,float *p);
void VecAddition(float *a,float *c,float *res);
void VecSubtraction(float *a,float *c,float *res,int width);
void VecScalAddition(float *a,float c,int size);
void MatScalMul(float *a,float m,int rows);
void MatTranspose(float *a,float *at,int width,int height);
float VecSum(float *a);
void predict(float *datos,float *pesos,float bias,float *preds,int h,int w);
float score(float *preds, float *y_r,int w);
int buscaMejorModelo(float *a);
void train_serial();
void train_paralelo();
void parallTranspose(float *a,float *at);
void parallMult(float *a,float *d,float *c,int ha,int wa,int wb);

float astep = alphamax/NUM_MODELOS;

// global timers
double serialTimer = 0.0;
float parallelTimer = 0.0;

float *X,*y,*W,*Jc,*XT,*y_pred,*Xtest,*ytest,*y_predtest,*modelos,*alphas, *d_X, *d_W, *d_XT, *d_Y, *d_Xtest, *d_modelos;
int sizeX=HX*WX*sizeof(float );
int sizeXtest=HXtest*WX*sizeof(float );
int sizeY=HY*sizeof(float );
int sizeYtest=HYtest*sizeof(float );
int sizeW=WX*sizeof(float );
int sizeJ=NUM_ITER* sizeof(float);
int sizeMod=NUM_MODELOS* sizeof(float);

__device__ float d_b, *d_S, *d_Sr, *d_cost, *d_dW, *d_Z, *d_Jc, *dZ,*dS;
__device__ float  * d_yPredTest, *d_ytest, *d_st, *d_yPred, *d_s;


// Kernel multilplicacion matrices hilo computa 1 elemento de C
__global__ void onelementMtxMult (float *d_a, float *d_d, float *d_c, int ha, int wa, int wb){
    int i = threadIdx.y + (blockIdx.y * blockDim.y);
    int j = threadIdx.x + (blockIdx.x * blockDim.x);
    float temp = 0;
    if (i < ha && j < wb){
        for (int k =0; k < wa; k++){
            temp += d_a [i * wa + k] * d_d [j + (wb * k)];
        }
        d_c [i * wb + j] = temp;
    }

}


__global__ void parallScaleAdd (float *d_Z, float d_b, int d_size){
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index < d_size){
        d_Z[index] = d_Z[index] + d_b;

    }
}
__global__ void parallSigmoid (float *d_Z, float *d_S, int d_size){
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index < d_size){
        d_S[index] = 1.0/(1 + exp(-d_Z[index]));
    }
}

__global__ void parallCost ( float *d_S, float *d_Y){
    float temp =0;
    for (int index = 0 ; index < HY ; index ++) {
        temp += d_Y[index] * log(d_S[index]) + (1 - d_Y[index]) * log(1 - d_S[index]);
    }
    *d_cost = float(-(1.0/HY)) * temp;

    }



__global__ void parallSub (float *d_a,float *d_c,float *d_res,int d_width){
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index < d_width){
        d_res [index] = d_a [index] - d_c [index];

    }
}

__global__ void parallScaleMult (float *d_a,float d_m,int d_rows){
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index < d_rows){
        d_a [index] = d_a [index] * d_m;

    }
}

__global__ void parallBin (float *dS,float *preds,int size){
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index < size){
        if(dS [index] <= 0.5){
            preds[index] = 0;
        }
        else{
            preds[index] = 1;
        }

    }
}

__global__ void parallPredict (float *datos,float *pesos,float bias,float *preds,int h,int w){
    dZ = new float [h];
    dS = new float [h];
    dim3 block(32, 32);
    auto gridx =  1 / block.x + 1;
    auto gridy =  h / block.y + 1;
    dim3 grid(gridx, gridy);
    onelementMtxMult<<<grid, block>>>(datos, pesos, dZ, h, w, 1 );

    int b = ceil((float)HY / 1024.0) + 1;
    int t = 1024;
    parallScaleAdd <<<b, t >>>( dZ, bias, HY);

    b = ceil((float)h / 1024.0) + 1;
    parallSigmoid <<<b, t >>>( dZ, dS, h);

    parallBin <<<b, t>>> (dS, preds, h);

    delete(dZ),delete(dS);

}


__global__ void parallScore(float *point, float *preds, float *y_r,int w){
    float temp = 0;
    for(int i=0;i<w;i++){
        if(preds[i]==y_r[i]){
            temp += 1;
        }
    }
    *point =(temp/float (w))*100;
}

//Kernel padre para entrenamiento de los modelos
__global__ void parallTrain(float *d_X,float *d_XT, float *d_W, float *d_Y, float *d_Xtest, float *d_ytest ,float *d_modelos){
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    d_yPredTest = new float [HYtest];
    d_yPred =  new float [HY];
    float astep = alphamax/NUM_MODELOS;
    if (index < NUM_MODELOS){
            d_b = 0;
            d_Jc = new float [NUM_ITER];
            for (int iter = 0; iter < NUM_ITER; iter++) {
                d_S = new float[HY];
                d_Sr = new float[HY];
                d_dW = new float[WX];
                d_Z = new float[HY];
                dim3 block(32, 32);
                auto gridx =  1 / block.x + 1;
                auto gridy =  HX / block.y + 1;
                dim3 grid(gridx, gridy);
                onelementMtxMult<<<grid, block>>>(d_X, d_W, d_Z, HX, WX, 1);
                int b = ceil((float)HY / 1024.0) + 1;
                int t = 1024;
                parallScaleAdd <<<b, t >>>( d_Z, d_b, HY);

                parallSigmoid <<<b, t >>>( d_Z, d_S, HY);

                parallCost <<<1, 1 >>> (d_S, d_Y);
                d_Jc [iter] = *d_cost;

                parallSub <<<b, t >>> (d_S, d_Y, d_Sr, HY);

                gridx =  1 / block.x + 1;
                gridy =  WX / block.y + 1;
                dim3 grid2(gridx,gridy);
                onelementMtxMult<<<grid2, block>>>(d_XT, d_Sr, d_dW, WX, HX, 1);

                b = ceil((float)WX / 1024.0) + 1;
                parallScaleMult <<<b, t >>> (d_dW, 1.0/ HY, WX);

                float db = 0;
                for(int ii= 0; ii<HY; ii++) {
                    db += d_Sr[ii];}
                db = db/ HY;

                parallScaleMult <<<b, t >>> (d_dW, (float) (astep * index) , WX);
                db = (float) (astep * index) * db;

                parallSub <<<b, t >>> (d_W, d_dW, d_W, WX);

                d_b = d_b - db;
                delete(d_dW), delete(d_S), delete(d_Sr), delete(d_Z);

            }
            //PREDICCION

        //childOptimizer <<<b, t >>>( d_X, d_XT, d_W);
        hipDeviceSynchronize();

        parallPredict <<< 1, 1 >>> (d_Xtest, d_W, d_b, d_yPredTest, HXtest, WX);

        parallScore<<<1, 1>>>(d_st,d_yPredTest, d_ytest, HYtest);

        parallPredict <<< 1, 1 >>> (d_X, d_W, d_b, d_yPred, HX, WX);

        parallScore<<<1, 1>>>(d_s, d_yPred, d_Y, HY);

        d_modelos[index] = *d_st;

    }


}





// Kernel MatrixTranspose
__global__ void PTranspose(float *d_a,float *d_at){
    int i = threadIdx.y + (blockIdx.y * blockDim.y);
    int j = threadIdx.x + (blockIdx.x * blockDim.x);
    if (i<HX && j<WX){
        d_at[j*HX+i] = d_a[i*WX+j];

    }
}



int main() {
    //Datos para entrenar
    X = (float *) malloc(sizeX);
    XT = (float *) malloc(sizeX);
    y = (float *) malloc(sizeY);
    y_pred = (float *) malloc(sizeY);
    y_predtest = (float *) malloc(sizeYtest);
    W = (float *) malloc(sizeW);
    Jc = (float *) malloc(sizeJ);
    //Datos de test
    Xtest = (float *) malloc(sizeXtest);
    ytest = (float *) malloc(sizeYtest);
    //Datos de los modelos
    modelos = (float *) malloc(sizeMod);
    alphas = (float *) malloc(sizeMod);

    leerDatos(X,y,FILE_NAME_TRAIN);
    leerDatos(Xtest,ytest,FILE_NAME_TEST);



//Proceso Paralelo
    train_paralelo();

    y_pred = (float *) malloc(sizeY);
    y_predtest = (float *) malloc(sizeYtest);
    W = (float *) malloc(sizeW);
    Jc = (float *) malloc(sizeJ);

    //Datos de los modelos
    modelos = (float *) malloc(sizeMod);
    alphas = (float *) malloc(sizeMod);

    //Proceso serial
    clock_t start = clock();
    train_serial();
    clock_t end = clock();
    serialTimer = double (end-start) / double(CLOCKS_PER_SEC);
    cout << "Tiempo del proceso serial: " << serialTimer << endl;


    free(X), free(y); free(W),free(y_pred),free(Xtest),free(ytest),free(y_predtest);
    return 0;
}

void MatMul(float *a,float *d,float *c,int ha,int wa,int wb){
    float sum;
    for(int i= 0; i<ha; i++) {
        for(int j=0;j<wb; j++){
            sum = 0;
            for(int k=0;k<wa; k++){
                sum+=a[i*wa+k]*d[j+(wb*k)];
            }
            c[i*wb+j] = sum;
        }
    }
}


void printMatrix(float *h_a, int rows, int cols){
    for(int i=0; i<rows; i++ ){
        for(int j=0; j<cols; j++){
            cout<<h_a[i*cols+j]<<" ";
        }
        cout<<endl;
    }
}

void generateRandom(float *h_a,int rows, int cols){
    // Initialize seed
    srand(time(NULL));
    for(int i=0; i<rows*cols; i++){
        h_a[i] = rand() % 10 + 1;
    }
}

void leerDatos(float *X,float *y,string file){
    ifstream archivo(file);
    string linea;
    char delimiter = ',';
    getline(archivo,linea);
    int i=0;
    while (getline(archivo,linea)){
        stringstream stream(linea);
        string num,code,grosor_tumor, tam_celula, form_celula, adhesion, celula_epit, nucleos,cromatina,nucleos_normales,mitosis,clase;

        getline(stream,num,delimiter);
        //getline(stream,code,delimiter);
        getline(stream,grosor_tumor,delimiter);
        getline(stream,tam_celula,delimiter);
        getline(stream,form_celula,delimiter);
        getline(stream,adhesion,delimiter);
        getline(stream,celula_epit,delimiter);
        getline(stream,nucleos,delimiter);
        getline(stream,cromatina,delimiter);
        getline(stream,nucleos_normales,delimiter);
        getline(stream,mitosis,delimiter);
        getline(stream,clase,delimiter);

        X[i*WX] = stof(grosor_tumor);
        X[i*WX+1] = stof(tam_celula);
        X[i*WX+2] = stof(form_celula);
        X[i*WX+3] = stof(adhesion);
        X[i*WX+4] = stof(celula_epit);
        X[i*WX+5] = stof(nucleos);
        X[i*WX+6] = stof(cromatina);
        X[i*WX+7] = stof(nucleos_normales);
        X[i*WX+8] = stof(mitosis);
        y[i] = stof(clase);

        i++;
    }
    archivo.close();
}

void sigmoid(float *a,float *c,int h){
    for(int i=0;i<h;i++){
        c[i] = 1/(1+exp(-a[i]));

    }
}

float costoCE(float *a,float *p){
    float suma=0;
    for(int i=0;i<HY;i++){
        suma +=  p[i]*log(a[i])+(1-p[i])*log(1-a[i]);
    }
    return float (-(1.0/HY))*suma;
}

void VecAddition(float *a,float *c,float *res){
    for(int i= 0; i<HY; i++) {
        res[i] = a[i] + c[i];
    }
}
void VecScalAddition(float *a,float c,int size){
    for(int i= 0; i<size; i++) {
        a[i] = a[i] + c;
    }
}

float VecSum(float *a){
    float suma=0;
    for(int i= 0; i<HY; i++) {
        suma += a[i];
    }
    return suma;
}

void VecSubtraction(float *a,float *c,float *res,int width){
    for(int i= 0; i<width; i++) {
        res[i] = a[i] - c[i];
    }
}
void MatScalMul(float *a,float m,int rows){
    for (int i=0;i<rows;i++){
        a[i] = a[i]*m;
    }
}

void MatTranspose(float *a,float *at,int width,int height){
    for(int j=0;j<width;j++){
        for(int i=0;i<height;i++){
            at[j*height+i] = a[i*width+j];
        }
    }
}

void predict(float *datos,float *pesos,float bias,float *preds,int h,int w){
    float *Z,*S;
    Z = (float *) malloc(h*sizeof(float ));
    S = (float *) malloc(h*sizeof(float ));
    MatMul(datos,pesos,Z,h,w,1);
    VecScalAddition(Z,bias,h);
    sigmoid(Z,S,h);
    for(int i=0;i<h;i++){
        if(S[i]<=0.5){
            preds[i] = 0;
        }
        else{
            preds[i] = 1;
        }
    }
    free(Z),free(S);
}

float score(float *preds, float *y_r,int w){
    float aciertos = 0;
    for(int i=0;i<w;i++){
        if(preds[i]==y_r[i]){
            aciertos+=1;
        }
    }
    return (aciertos/float (w))*100;
}

void generarAlphas(float *a,float step){
    for(int i=0;i<NUM_MODELOS;i++){
        a[i] = float (i)*step;
    }
}

int buscaMejorModelo(float *a){
    float maximo = a[0];
    int maxindex=0;
    for(int i=1;i<NUM_MODELOS;i++){
        if(a[i]>maximo){
            maximo = a[i];
            maxindex = i;
        }
    }
    return maxindex;
}

void train_serial() {
    MatTranspose(X,XT,WX,HX);
    generarAlphas(alphas, astep);
    for (int model = 0; model < NUM_MODELOS; model++) {
        float b = 0;
        for (int iter = 0; iter < NUM_ITER; iter++) {
            float *S, *Sr, cost, *dW, *Z;
            S = (float *) malloc(sizeY);
            Sr = (float *) malloc(sizeY);
            dW = (float *) malloc(sizeW);
            Z = (float *) malloc(sizeY);

            MatMul(X, W, Z, HX, WX, 1);
            VecScalAddition(Z, b, HY);
            sigmoid(Z, S, HY);
            cost = costoCE(S, y);
            //cout << "Iteración " << iter + 1 << " : " << cost << endl;
            Jc[iter] = cost;
            VecSubtraction(S, y, Sr, HY);
            MatMul(XT, Sr, dW, WX, HX, 1);
            MatScalMul(dW, 1.0 / HY, WX);
            float db = VecSum(Sr);
            db = db / HY;

            MatScalMul(dW, alphas[model], WX);
            db = alphas[model] * db;

            VecSubtraction(W, dW, W, WX);
            b = b - db;

            free(dW), free(S), free(Sr), free(Z);
        }
        predict(Xtest, W, b, y_predtest, HXtest, WX);
        float st = score(y_predtest, ytest, HYtest);
        //cout << "Score del modelo test: " << st << "%" << endl;
        predict(X, W, b, y_pred, HX, WX);
        float s = score(y_pred, y, HY);
        //cout << "Score del modelo entrenamiento: " << s << "%" << endl;
        modelos[model] = st;
    }
    int index = buscaMejorModelo(modelos);
    cout << "Mejor score obtenido: " << modelos[index] << endl;
    cout << "Alpha del mejor modelo: " << alphas[index] << endl;
}

void train_paralelo() {

    clock_t start = clock();
    parallTranspose(X,XT);
    hipMalloc((void **)&d_X, sizeX);
    hipMalloc((void **)&d_XT, sizeX);
    hipMalloc((void **)&d_W, sizeW);
    hipMalloc((void **)&d_Y, sizeY);
    hipMalloc((void **)&d_Xtest, sizeXtest);
    hipMalloc((void **)&d_ytest, sizeYtest);
    hipMalloc((void **)&d_modelos, sizeMod);



    // Transferir datos de host a device
    hipMemcpy(d_X, X, sizeX, hipMemcpyHostToDevice);
    hipMemcpy(d_XT, XT, sizeX, hipMemcpyHostToDevice);
    hipMemcpy(d_W, W, sizeW, hipMemcpyHostToDevice);
    hipMemcpy(d_Y, y, sizeY, hipMemcpyHostToDevice);
    hipMemcpy(d_Xtest, Xtest, sizeXtest, hipMemcpyHostToDevice);
    hipMemcpy(d_ytest, ytest, sizeYtest, hipMemcpyHostToDevice);
    hipMemcpy(d_modelos, modelos, sizeMod, hipMemcpyHostToDevice);


    int blocks = ceil(NUM_MODELOS / 1024) + 1;
    int threads = 1024;

    parallTrain<<<blocks, threads>>>(d_X,d_XT, d_W,d_Y,d_Xtest, d_ytest, d_modelos);
    hipMemcpy(modelos, d_modelos, sizeMod, hipMemcpyDeviceToHost);


    int index = buscaMejorModelo(modelos);
    cout << "Mejor score obtenido: " << modelos[index] << endl;
    cout << "Alpha del mejor modelo: " << alphas[index] << endl;

    clock_t end = clock();
    parallelTimer = double (end-start) / double(CLOCKS_PER_SEC);
    cout << "Tiempo del proceso paralelo: " << parallelTimer << endl;
}



void parallTranspose(float *a,float *at){
    float * d_a, * d_at;
    hipMalloc((void **)&d_a, sizeX);
    hipMalloc((void **)&d_at, sizeX);

    hipMemcpy(d_a, a, sizeX, hipMemcpyHostToDevice);
    hipMemcpy(d_at, at, sizeX, hipMemcpyHostToDevice);


    dim3 block(32, 32);
    auto gridx = WX/block.x +1;
    auto gridy = HX/block.y +1;

    dim3 grid(gridx,gridy );


    PTranspose<<<grid, block>>>(d_a, d_at);
    // hipDeviceSynchronize ();



    // Copy data from device to host
    hipMemcpy(at, d_at, sizeX, hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree (d_at);
/*
    for (int i =0; i< sizeX ; i++){
        cout << a[i]<< " ";
    }
    cout <<endl;
    for (int i =0; i< sizeX ; i++){
        cout << at[i]<< " ";
    }
    cout <<endl;*/


}


