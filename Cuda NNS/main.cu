#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include<fstream>
#include <sstream>
#include <vector>
#define HX 558 //Filas de X  train:558, total:699
#define WX 9 //Columnas de X
#define HY 558 //Filas de Y train:558, total:699
#define HXtest 141
#define HYtest 141
#define FILE_NAME_TRAIN "/home/icasasola/proyecto2/breast-cancer-train.csv"
#define FILE_NAME_TEST "/home/icasasola/proyecto2/breast-cancer-test.csv"
#define NUM_ITER 2000
#define alphamax 0.5
#define NUM_MODELOS 500
#define NUM_HILOS 25
#define TILE_W 10 //Tamaño mosaicos
#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__));}
using namespace std;

void MatMul(float *a,float *d,float *c,int ha,int wa,int wb);
void generateRandom(float *h_a,int rows, int cols);
void compareVectors(float *parallel, float *serial, int rows, int cols);
void leerDatos(float *X,float *y,string file);
void printMatrix(float *h_a, int rows, int cols);
void sigmoid(float *a,float *c, int w,int h);
float costoCE(float *a,float *p);
void VecAddition(float *a,float *c,float *res);
void VecSubtraction(float *a,float *c,float *res,int width);
void VecScalAddition(float *a,float c,int size);
void MatScalMul(float *a,float m,int rows);
void MatTranspose(float *a,float *at,int width,int height);
float VecSum(float *a);
void predict(float *datos,float *pesos,float bias,float *preds,int h,int w);
float score(float *preds, float *y_r,int w);
void generarAlphas(float *a,float step);
int buscaMejorModelo(float *a);
void train_serial();
void train_parallel();

//Salto de alpha
float astep = alphamax/NUM_MODELOS;

// global timers
double serialTimer = 0.0;
float parallelTimer = 0.0;

float *X,*y,*W,*Jc,*XT,*y_pred,*Xtest,*ytest,*y_predtest,*modelos,*alphas,*modelos_par;
float *d_mod,*d_alps,*d_X,*d_W,*d_XT,*d_y,*d_Xtest,*d_ytest;
int sizeX=HX*WX*sizeof(float );
int sizeXtest=HXtest*WX*sizeof(float );
int sizeY=HY*sizeof(float );
int sizeYtest=HYtest*sizeof(float );
int sizeW=WX*sizeof(float );
int sizeJ=NUM_ITER* sizeof(float);
int sizeMod=NUM_MODELOS* sizeof(float);


__device__ void MatMulPar(float *a,float *d,float *c,int ha,int wa,int wb){
    float sum;
    for(int i= 0; i<ha; i++) {
        for(int j=0;j<wb; j++){
           sum = 0;
           for(int k=0;k<wa; k++){
               sum+=a[i*wa+k]*d[j+(wb*k)];
           }
           c[i*wb+j] = sum;
        }
    }
}

__device__ void sigmoidPar(float *a,float *c, int w,int h){
    for(int i=0;i<h;i++){
        for(int j=0;j<w;j++){
            c[i*w+j] = 1/(1+exp(-a[i*w+j]));
        }
    }
}

__device__ float costoCEPar(float *a,float *p){
    float suma=0;
    for(int i=0;i<HY;i++){
        suma +=  p[i]*log(a[i])+(1-p[i])*log(1-a[i]);
    }
    return float (-(1.0/HY))*suma;
}

__device__ void VecScalAdditionPar(float *a,float c,int size){
    for(int i= 0; i<size; i++) {
            a[i] = a[i] + c;
    }
}

__device__ float VecSumPar(float *a){
    float suma=0;
    for(int i= 0; i<HY; i++) {
            suma += a[i];
    }
    return suma;
}

__device__ void VecSubtractionPar(float *a,float *c,float *res,int width){
    for(int i= 0; i<width; i++) {
            res[i] = a[i] - c[i];
    }
}

__device__ void MatScalMulPar(float *a,float m,int rows){
    for (int i=0;i<rows;i++){
        a[i] = a[i]*m;
    }
}

__device__ void predictPar(float *datos,float *pesos,float bias,float *preds,int h,int w){
    float *Z,*S;
    Z = (float *) malloc(h*sizeof(float ));
    S = (float *) malloc(h*sizeof(float ));
    MatMulPar(datos,pesos,Z,h,w,1);
    VecScalAdditionPar(Z,bias,h);
    sigmoidPar(Z,S,1,h);
    for(int i=0;i<h;i++){
        if(S[i]<=0.5){
            preds[i] = 0;
        }
        else{
            preds[i] = 1;
        }
    }
    free(Z),free(S);
}

__device__ float scorePar(float *preds, float *y_r,int w){
    float aciertos = 0;
    for(int i=0;i<w;i++){
        if(preds[i]==y_r[i]){
            aciertos+=1;
        }
    }
    return (aciertos/w)*100;
}

__device__ int buscaMejorModeloPar(float *a){
    float maximo = a[0];
    int maxindex=0;
    for(int i=1;i<NUM_MODELOS;i++){
        if(a[i]>maximo){
            maximo = a[i];
            maxindex = i;
        }
    }
    return maxindex;
}

__global__ void train_model(float *mod, float *alps, float *dat, float *datT,float *label,float *datTest,float *labelTest) {

        int model = threadIdx.x + (blockIdx.x * blockDim.x);
        if (model<NUM_MODELOS) {
            float b = 0;
            float *WP;
            float *label_pred;
            label_pred = (float *) malloc(HYtest * sizeof(float));
            WP = (float *) malloc(WX * sizeof(float));

            for (int iter = 0; iter < NUM_ITER; iter++) {
                float *S, *Sr, cost, *dW, *Z;
                S = (float *) malloc(HY * sizeof(float));
                Sr = (float *) malloc(HY * sizeof(float));
                dW = (float *) malloc(WX * sizeof(float));
                Z = (float *) malloc(HY * sizeof(float));

                MatMulPar(dat, WP, Z, HX, WX, 1);
                VecScalAdditionPar(Z, b, HY);
                sigmoidPar(Z, S, 1, HY);
                cost = costoCEPar(S, label);
                //cout << "Iteración " << iter + 1 << " : " << cost << endl;
                //Jc[iter] = cost;
                VecSubtractionPar(S, label, Sr, HY);
                MatMulPar(datT, Sr, dW, WX, HX, 1);
                MatScalMulPar(dW, 1.0 / HY, WX);
                float db = VecSumPar(Sr);
                db = db / HY;

                MatScalMulPar(dW, alps[model], WX);
                db = alps[model] * db;

                VecSubtractionPar(WP, dW, WP, WX);
                b = b - db;

                free(dW), free(S), free(Sr), free(Z);
            }
            predictPar(datTest, WP, b, label_pred, HXtest, WX);
            float st = scorePar(label_pred, labelTest, HYtest);

            mod[model] = st;
            free(label_pred);
            free(WP);
        }
}

int main() {
    //Datos para entrenar
    X = (float *) malloc(sizeX);
    XT = (float *) malloc(sizeX);
    y = (float *) malloc(sizeY);
    y_pred = (float *) malloc(sizeY);
    y_predtest = (float *) malloc(sizeYtest);
    W = (float *) malloc(sizeW);
    Jc = (float *) malloc(sizeJ);
    //Datos de test
    Xtest = (float *) malloc(sizeXtest);
    ytest = (float *) malloc(sizeYtest);
    //Datos de los modelos
    modelos = (float *) malloc(sizeMod);
    alphas = (float *) malloc(sizeMod);
    modelos_par = (float *) malloc(sizeMod);

    //Reservar memoria en device
    hipMalloc((void **)&d_X, sizeX);
    hipMalloc((void **)&d_Xtest, sizeX);
    hipMalloc((void **)&d_y, sizeY);
    hipMalloc((void **)&d_ytest, sizeY);
    hipMalloc((void **)&d_XT, sizeX);
    hipMalloc((void **)&d_mod, sizeMod);
    hipMalloc((void **)&d_alps, sizeMod);

    leerDatos(X,y,FILE_NAME_TRAIN);
    leerDatos(Xtest,ytest,FILE_NAME_TEST);
    MatTranspose(X,XT,WX,HX);

    generarAlphas(alphas, astep);

    // Transferir datos de host a device
    hipMemcpy(d_alps, alphas, sizeMod, hipMemcpyHostToDevice);
    hipMemcpy(d_X, X, sizeX, hipMemcpyHostToDevice);
    hipMemcpy(d_Xtest, Xtest, sizeX, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeY, hipMemcpyHostToDevice);
    hipMemcpy(d_ytest, ytest, sizeY, hipMemcpyHostToDevice);
    hipMemcpy(d_XT, XT, sizeX, hipMemcpyHostToDevice);

    //Proceso serial
    clock_t start = clock();
    train_serial();
    clock_t end = clock();
    serialTimer = double (end-start) / double(CLOCKS_PER_SEC);
    cout << "Tiempo del proceso serial: " << serialTimer << endl;

    //Proceso en paralelo
    train_parallel();
    //compareVectors(modelos_par,modelos,NUM_MODELOS,1);

    cout << "Speed-up: " << serialTimer / (parallelTimer /1000)<< "X"<<endl;
    cout << "\n"<<endl;

    hipFree(d_alps),hipFree(d_X),hipFree(d_Xtest),hipFree(d_y),hipFree(d_ytest);
    hipFree(d_XT), hipFree(d_mod);
    free(X), free(y); free(W),free(y_pred),free(Xtest),free(ytest),free(y_predtest);
    free(modelos),free(alphas);
    return 0;
}

void MatMul(float *a,float *d,float *c,int ha,int wa,int wb){
    float sum;
    for(int i= 0; i<ha; i++) {
        for(int j=0;j<wb; j++){
           sum = 0;
           for(int k=0;k<wa; k++){
               sum+=a[i*wa+k]*d[j+(wb*k)];
           }
           c[i*wb+j] = sum;
        }
    }
}

void printMatrix(float *h_a, int rows, int cols){
    for(int i=0; i<rows; i++ ){
        for(int j=0; j<cols; j++){
            cout<<h_a[i*cols+j]<<" ";
        }
        cout<<endl;
    }
}

void generateRandom(float *h_a,int rows, int cols){
    // Initialize seed
    srand(time(NULL));
    for(int i=0; i<rows*cols; i++){
        h_a[i] = rand() % 10 + 1;
    }
}

void compareVectors(float *parallel, float *serial, int rows, int cols){

    int diff = 0;
    for(int i= 0; i<rows*cols; i++) {
        if (parallel[i] != serial[i]) {
            diff++;
            //cout<<i<<". "<<parallel[i] << " " << serial[i] << "\n" << endl;
        }
    }

    if(diff>0){
        cout<< diff <<" elements different" << endl;
    }
    else
        cout << "Vectors are equal!..." << endl;
}

void leerDatos(float *X,float *y,string file){
    ifstream archivo(file);
    string linea;
    char delimiter = ',';
    getline(archivo,linea);
    int i=0;
    while (getline(archivo,linea)){
        stringstream stream(linea);
        string num,code,grosor_tumor, tam_celula, form_celula, adhesion, celula_epit, nucleos,cromatina,nucleos_normales,mitosis,clase;

        getline(stream,num,delimiter);
        //getline(stream,code,delimiter);
        getline(stream,grosor_tumor,delimiter);
        getline(stream,tam_celula,delimiter);
        getline(stream,form_celula,delimiter);
        getline(stream,adhesion,delimiter);
        getline(stream,celula_epit,delimiter);
        getline(stream,nucleos,delimiter);
        getline(stream,cromatina,delimiter);
        getline(stream,nucleos_normales,delimiter);
        getline(stream,mitosis,delimiter);
        getline(stream,clase,delimiter);

        X[i*WX] = stof(grosor_tumor);
        X[i*WX+1] = stof(tam_celula);
        X[i*WX+2] = stof(form_celula);
        X[i*WX+3] = stof(adhesion);
        X[i*WX+4] = stof(celula_epit);
        X[i*WX+5] = stof(nucleos);
        X[i*WX+6] = stof(cromatina);
        X[i*WX+7] = stof(nucleos_normales);
        X[i*WX+8] = stof(mitosis);
        y[i] = stof(clase);

//        cout<<"Fila: "<<i<<endl;
//        cout<<X[i*WX]<<" ";
//        cout<<X[i*WX+1]<<" ";
//        cout<<X[i*WX+2]<<" ";
//        cout<<X[i*WX+3]<<" ";
//        cout<<X[i*WX+4]<<" ";
//        cout<<X[i*WX+5]<<" ";
//        cout<<X[i*WX+6]<<" ";
//        cout<<X[i*WX+7]<<" ";
//        cout<<X[i*WX+8]<<" ";
//        cout<<y[i]<<endl;
//        cout<<"\n"<<endl;
        i++;
    }
    archivo.close();
}

void sigmoid(float *a,float *c, int w,int h){
    for(int i=0;i<h;i++){
        for(int j=0;j<w;j++){
            c[i*w+j] = 1/(1+exp(-a[i*w+j]));
        }
    }
}

float costoCE(float *a,float *p){
    float suma=0;
    for(int i=0;i<HY;i++){
        suma +=  p[i]*log(a[i])+(1-p[i])*log(1-a[i]);
    }
    return float (-(1.0/HY))*suma;
}

void VecAddition(float *a,float *c,float *res){
    for(int i= 0; i<HY; i++) {
            res[i] = a[i] + c[i];
    }
}
void VecScalAddition(float *a,float c,int size){
    for(int i= 0; i<size; i++) {
            a[i] = a[i] + c;
    }
}

float VecSum(float *a){
    float suma=0;
    for(int i= 0; i<HY; i++) {
            suma += a[i];
    }
    return suma;
}

void VecSubtraction(float *a,float *c,float *res,int width){
    for(int i= 0; i<width; i++) {
            res[i] = a[i] - c[i];
    }
}
void MatScalMul(float *a,float m,int rows){
    for (int i=0;i<rows;i++){
        a[i] = a[i]*m;
    }
}

void MatTranspose(float *a,float *at,int width,int height){
    for(int j=0;j<width;j++){
        for(int i=0;i<height;i++){
            at[j*height+i] = a[i*width+j];
        }
    }
}

void predict(float *datos,float *pesos,float bias,float *preds,int h,int w){
    float *Z,*S;
    Z = (float *) malloc(h*sizeof(float ));
    S = (float *) malloc(h*sizeof(float ));
    MatMul(datos,pesos,Z,h,w,1);
    VecScalAddition(Z,bias,h);
    sigmoid(Z,S,1,h);
    for(int i=0;i<h;i++){
        if(S[i]<=0.5){
            preds[i] = 0;
        }
        else{
            preds[i] = 1;
        }
    }
    free(Z),free(S);
}

float score(float *preds, float *y_r,int w){
    float aciertos = 0;
    for(int i=0;i<w;i++){
        if(preds[i]==y_r[i]){
            aciertos+=1;
        }
    }
    return (aciertos/float (w))*100;
}

void generarAlphas(float *a,float step){
    for(int i=0;i<NUM_MODELOS;i++){
        a[i] = float (i+1)*step;
    }
}

int buscaMejorModelo(float *a){
    float maximo = a[0];
    int maxindex=0;
    for(int i=1;i<NUM_MODELOS;i++){
        if(a[i]>maximo){
            maximo = a[i];
            maxindex = i;
        }
    }
    return maxindex;
}

void train_serial() {
    for (int model = 0; model < NUM_MODELOS; model++) {
        float b = 0;
        for (int iter = 0; iter < NUM_ITER; iter++) {
            float *S, *Sr, cost, *dW, *Z;
            S = (float *) malloc(sizeY);
            Sr = (float *) malloc(sizeY);
            dW = (float *) malloc(sizeW);
            Z = (float *) malloc(sizeY);

            MatMul(X, W, Z, HX, WX, 1);
            VecScalAddition(Z, b, HY);
            sigmoid(Z, S, 1, HY);
            cost = costoCE(S, y);
            //cout << "Iteración " << iter + 1 << " : " << cost << endl;
            Jc[iter] = cost;
            VecSubtraction(S, y, Sr, HY);
            MatMul(XT, Sr, dW, WX, HX, 1);
            MatScalMul(dW, 1.0 / HY, WX);
            float db = VecSum(Sr);
            db = db / HY;

            MatScalMul(dW, alphas[model], WX);
            db = alphas[model] * db;

            VecSubtraction(W, dW, W, WX);
            b = b - db;

            free(dW), free(S), free(Sr), free(Z);
        }
        predict(Xtest, W, b, y_predtest, HXtest, WX);
        float st = score(y_predtest, ytest, HYtest);
        //cout << "Score del modelo test: " << st << "%" << endl;
        predict(X, W, b, y_pred, HX, WX);
        float s = score(y_pred, y, HY);
        //cout << "Score del modelo entrenamiento: " << s << "%" << endl;
        modelos[model] = st;
    }
    //cout<<"Costo:"<<endl;
    //printMatrix(Jc,NUM_ITER,1);
    int index = buscaMejorModelo(modelos);
    cout<<"Mejor score obtenido: "<<modelos[index]<<endl;
    cout<<"Alpha del mejor modelo: "<<alphas[index]<<endl;
}

void train_parallel(){

    int blocks = ceil(NUM_MODELOS / NUM_HILOS) + 1;

    // define timers
    hipEvent_t start, stop;

    // events to take time
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);
    CUDA_CALL( hipGetLastError() );
    // Launch kernel
    train_model<<<blocks, NUM_HILOS>>>(d_mod,d_alps,d_X,d_XT,d_y,d_Xtest,d_ytest);
    CUDA_CALL( hipGetLastError() );

    hipEventRecord(stop,0);

    hipEventSynchronize(stop);
    hipMemcpy(modelos_par, d_mod, sizeMod, hipMemcpyDeviceToHost);
    int index = buscaMejorModelo(modelos_par);
    cout<<"Mejor score obtenido: "<<modelos_par[index]<<endl;
    cout<<"Alpha del mejor modelo: "<<alphas[index]<<endl;

    hipEventElapsedTime(&parallelTimer, start, stop);

    cout<< "Tiempo del proceso en paralelo: " << parallelTimer << " ms, " << parallelTimer / 1000 << " secs" <<endl;

    // Copy data from device to host
    //hipMemcpy(modelos_par, d_mod, sizeMod, hipMemcpyDeviceToHost);
}